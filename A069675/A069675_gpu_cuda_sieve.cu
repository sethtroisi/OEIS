#include <iostream>

#include "A069675_gpu_shared.h"

using namespace std;

#define checkCuda(expr) {                          \
  auto status = (expr);                             \
  if (status != hipSuccess) {                       \
    cerr << "cuda Error on line " << __LINE__ << ": " \
         << hipGetErrorString(status) << endl;        \
    exit(EXIT_FAILURE);                                 \
  }                                                      \
}


// TODO figure out what to set here
#define GRID_SIZE 16
#define BLOCK_SIZE 4

CUDAGLOBAL void FilterSieveKernel(
    long *is_prime_ref,
    long *div_mods,
    long *primes,
    long pi_start,
    long pi_end,
    bool *results) {

  // TODO LOTS of cuda calls
  int index = threadIdx.x + blockIdx.x * BLOCK_SIZE;

  long range = pi_end - pi_start;
  long bottom = range * index / (BLOCK_SIZE * GRID_SIZE);
  long top = range * (index + 1) / (BLOCK_SIZE * GRID_SIZE);
  bottom += pi_start;
  top += pi_start;

  for (long pi = bottom; pi < top; pi++) {
    results[pi] = test_p(is_prime_ref, primes[pi], div_mods + 24 * pi);
  }
}

void FilterSieveKernelHost(
    long is_prime_ref[MAX_DIGITS_P1][10][10],
    long div_mods[][24],
    long primes[],
    long pi_start,
    long pi_end,
    bool results[]) {

  // Probably beneficial to compress 100 bits vs 6400
  long *d_is_prime = NULL;
  checkCuda(hipMalloc(&d_is_prime, sizeof(long) * MAX_DIGITS_P1 * 10 * 10));
  checkCuda(hipMemcpy(d_is_prime, (long*)is_prime_ref, sizeof(long) * MAX_DIGITS_P1 * 10 * 10, hipMemcpyHostToDevice));

  long *d_div_mods = NULL;
  checkCuda(hipMalloc(&d_div_mods, sizeof(long) * 24 * pi_end));
  checkCuda(hipMemcpy(d_div_mods, div_mods, sizeof(long) * 24 * pi_end, hipMemcpyHostToDevice));

  long *d_primes = NULL;
  checkCuda(hipMalloc(&d_primes, sizeof(long) * pi_end));
  checkCuda(hipMemcpy(d_primes, primes, sizeof(long) * pi_end, hipMemcpyHostToDevice));

  bool *d_results = NULL;
  checkCuda(hipMalloc(&d_results, pi_end));
  checkCuda(hipMemset(d_results, 0, pi_end));

  FilterSieveKernel<<<GRID_SIZE, BLOCK_SIZE>>>(
      d_is_prime, d_div_mods, d_primes, pi_start, pi_end, d_results);

  checkCuda(hipMemcpy(results, d_results, pi_end, hipMemcpyDeviceToHost));

  hipFree(d_is_prime);
  hipFree(d_div_mods);
  hipFree(d_primes);
  hipFree(d_results);
}

